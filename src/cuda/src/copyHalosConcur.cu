//                             copyHaloAsync.cu
//                             --------------------

/* Date:   6-Jun-2017
   Author: M. Rheinhardt
   Description: Copier functions for the different "plates" of the halo and th efull inner data cube with host-device concurrency.
                Load balance yet to be established.
*/

//C libraries

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Headers
//#include "defines.h"

extern int mx, my, mz, nx, ny, nz, nghost, iproc;
static long mxy;

static hipStream_t strFront=NULL, strBack=NULL, strBot=NULL, strTop=NULL, strLeftRight=NULL;
static long halo_yz_size;
static float *halo_yz, *d_halo_yz; 

/****************************************************************************************************************/
__host__ void initializeCopying()
{ 
        mxy=mx*my;
        halo_yz_size=2*nghost*ny*nz*sizeof(float);      // size of buffer for yz halos

        hipMalloc(&d_halo_yz,halo_yz_size);            // buffer for yz halos in device
        halo_yz=(float*) malloc(halo_yz_size);          // buffer for yz halos in host
 
        hipStreamCreate(&strFront);
        hipStreamCreate(&strBack);
        hipStreamCreate(&strBot);
        hipStreamCreate(&strTop);
        hipStreamCreate(&strLeftRight);
}
/****************************************************************************************************************/
__host__ void finalizeCopying()
{
        hipFree(&d_halo_yz);
        free(halo_yz);

        hipStreamDestroy(strFront);
        hipStreamDestroy(strBack);
        hipStreamDestroy(strBot);
        hipStreamDestroy(strTop);
        hipStreamDestroy(strLeftRight);
}
/****************************************************************************************************************/
__host__ void copyOxyPlates(float* grid, float* d_grid)
{
//  copies outer xy halos from host to device

        const long size=mxy*nghost*sizeof(float);
        const long offset=mxy*(mz-nghost);

        // front plate
        hipHostRegister(grid, size, hipHostRegisterDefault);
        hipMemcpyAsync(d_grid, grid, size, hipMemcpyHostToDevice, strFront);

        // back plate
        hipHostRegister(grid+offset, size, hipHostRegisterDefault);
        hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strBack);
}
/****************************************************************************************************************/
__host__ void copyOxzPlates(float* grid, float* d_grid)
{
//  copies outer xz halos from host to device

        const int size=mx*nghost*sizeof(float);

        long offset=mxy*nghost;
        int i;

        // bottom plate
        for (i=0;i<nz;i++)
        {
          hipHostRegister(grid+offset, size, hipHostRegisterDefault);
          hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strBot);
          offset+=mxy;
        }

        // top plate
        offset=mxy*nghost+mx*(my-nghost);
        for (i=0;i<nz;i++)
        {
          hipHostRegister(grid+offset, size, hipHostRegisterDefault);
          hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strTop);
          offset+=mxy;
        }
}
/****************************************************************************************************************/
__global__ void unpackOyzPlates(float* d_grid,float* d_halo_yz,int mx,int nx,int ny,int mxy,int nghost)
{
//  unpacks buffer for yz halos in global memory

        long halo_ind=threadIdx.x + (threadIdx.y + threadIdx.z*ny)*(2*nghost), grid_ind;
        const long start_offset=(mxy+mx)*nghost;

        grid_ind=start_offset + threadIdx.z*mxy + threadIdx.y*mx + threadIdx.x;
        if (threadIdx.x>=nghost) grid_ind+=nx;

        d_grid[grid_ind]=d_halo_yz[halo_ind];
}
/****************************************************************************************************************/
__host__ void copyOyzPlates(float* grid, float* d_grid)
{
//  copies outer yz halos from host to device: they are first packed into the buffer halo_yz, which is then copied 
//  into device buffer d_halo_yz, finally unpacked on device.

        const int size=nghost*sizeof(float);
        const int x_inc=mx-nghost;

        int i,j;
        long halo_ind=0;
        long offset=mx*(my+1)*nghost;

        for (i=0;i<nz;i++)
        {
                for (j=0;j<ny;j++)
                {
                        // left plate
                        hipMemcpy(halo_yz+halo_ind,grid+offset,size,hipMemcpyHostToHost);  // also async?
                        halo_ind+=nghost;
                        offset+=x_inc;
                        // right plate
                        hipMemcpy(halo_yz+halo_ind,grid+offset,size,hipMemcpyHostToHost);  // also async?
                        halo_ind+=nghost;
                        offset+=nghost;
                }
                offset+=2*mx*nghost;
        }
        hipHostRegister(halo_yz, halo_yz_size, hipHostRegisterDefault);
        hipMemcpyAsync(d_halo_yz, halo_yz, halo_yz_size, hipMemcpyHostToDevice, strLeftRight);

//  unpacking in global memory; done by GPU kernel in stream strLeftRight

        int numBlocks=1;
        dim3 threads(2*nghost,ny,nz);
        unpackOyzPlates<<<numBlocks,threads,0,strLeftRight>>>(d_grid,d_halo_yz,mx,nx,ny,mxy,nghost);
}
/****************************************************************************************************************/
__host__ void unlockHostMemOuter(float* grid,float* d_grid)
{
//  after copy of outer halos: synchronizes streams and releases pinned memory

     	// front and back plates
        hipStreamSynchronize(strFront);
	hipHostUnregister(grid);	

        hipStreamSynchronize(strBack);
	hipHostUnregister(grid+mxy*(mz-nghost));

        long offset=mxy*nghost;
        int i;

        // outer bottom plate
	hipStreamSynchronize(strBot);
        for (i=0;i<nz;i++)
        {
        	hipHostUnregister(grid+offset);
        	offset+=mxy;
        }
        // outer top plate
	hipStreamSynchronize(strTop);
        offset=mxy*nghost+mx*(my-nghost);
        for (i=0;i<nz;i++)
        {
        	hipHostUnregister(grid+offset);
        	offset+=mxy;
        }
	// left & right plates
        hipStreamSynchronize(strLeftRight);
        hipHostUnregister(halo_yz);
}
/****************************************************************************************************************/
__host__ void unlockHostMemInner(float* grid,float* d_grid)
{
//  after copy of inner halos: synchronizes streams and releases pinned memory

        long offset=(mxy+mx+1)*nghost;
        int i;

        hipStreamSynchronize(strFront);

        // inner front plate
        for (i=0;i<nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strBack);

        // inner back plate
        offset=mxy*nz+(mx+1)*nghost;
        for (i=0;i<nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strBot);

        // inner bottom plate
        offset=(2*mxy+mx+1)*nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strTop);

        // inner top plate
        offset=2*mxy*nghost+mx*ny+nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strLeftRight);
        hipHostUnregister(halo_yz);
}
/****************************************************************************************************************/
__host__ void copyOuterHalos(float* grid, float* d_grid)
{
//  copies complete outer halo

        copyOxyPlates(grid, d_grid);
        copyOxzPlates(grid, d_grid);
        copyOyzPlates(grid, d_grid);
 	unlockHostMemOuter(grid, d_grid);
}
/****************************************************************************************/
__host__ void copyIxyPlates(float* grid, float* d_grid)    // or kernel?
{
//  copies inner xy halos from device to host

        const size_t px=mx*sizeof(float);
        const size_t sx=nx*sizeof(float);

        long offset=(mxy+mx+1)*nghost;
        int i;

        // inner front plate
        for (i=0;i<nghost;i++)
        {
          hipHostRegister(grid+offset, px*ny, hipHostRegisterDefault);
          hipMemcpy2DAsync(grid+offset, px, d_grid+offset, px, sx, ny, hipMemcpyDeviceToHost, strFront);
          offset+=mxy;
        }
        // inner back plate
        offset=mxy*nz+(mx+1)*nghost;
        for (i=0;i<nghost;i++)
        {
          hipHostRegister(grid+offset, px*ny, hipHostRegisterDefault);
          hipMemcpy2DAsync(grid+offset, px, d_grid+offset, px, sx, ny, hipMemcpyDeviceToHost, strBack);
          offset+=mxy;
        }
}
/****************************************************************************************/
__host__ void copyIxzPlates(float* grid, float* d_grid)    // or __global__?
{
//  copies inner xz halos from device to host

        const int px=mx*sizeof(float);
        const int sx=nx*sizeof(float);

        int offset=(2*mxy+mx+1)*nghost;
        int i;

        // inner bottom plate
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostRegister(grid+offset, px*nghost, hipHostRegisterDefault);
          hipMemcpy2DAsync( grid+offset, px, d_grid+offset, px, sx, nghost, hipMemcpyDeviceToHost, strBot);
          offset+=mxy;
        }
        // inner top plate
        offset=2*mxy*nghost+mx*ny+nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostRegister(grid+offset, px*nghost, hipHostRegisterDefault);
          hipMemcpy2DAsync( grid+offset, px, d_grid+offset, px, sx, nghost, hipMemcpyDeviceToHost, strTop);
          offset+=mxy;
        }
}
/****************************************************************************************/
__global__ void packIyzPlates(float* d_grid,float* d_halo_yz,int mx,int nx,int ny,int mxy,int nghost)
{
//  packs inner yz halos in buffer d_halo_yz on device

        const long halo_ind=threadIdx.x + (threadIdx.y + threadIdx.z*(ny-2*nghost))*(2*nghost);
        const long start_offset=((mxy+mx)*2+1)*nghost;

        long grid_ind=start_offset + threadIdx.z*mxy + threadIdx.y*mx + threadIdx.x;
        if (threadIdx.x>=nghost) grid_ind+=nx-2*nghost;

  	d_halo_yz[halo_ind] = d_grid[grid_ind]; 
}
/****************************************************************************************/
__host__ void copyIyzPlates(float* grid, float* d_grid)
{
//  copies inner yz halos from device to host: they are first packed into the buffer d_halo_yz, which is then copied 
//  into host buffer halo_yz, finally unpacked on host.


        //d_halo_yz has to have at least size (2*nghost)*(ny-2*nghost)*(nz-2*nghost).
        const int size=nghost*sizeof(float);
        const long halo_size=2*nghost*(ny-2*nghost)*(nz-2*nghost)*sizeof(float);
        const int x_inc=nx-nghost;

        int i,j;
        int halo_ind=0;
        long offset=((mxy+mx)*2+1)*nghost;
        dim3 threads(2*nghost,ny-2*nghost,nz-2*nghost);

        packIyzPlates<<<1,threads,0,strLeftRight>>>(d_grid,d_halo_yz,mx,nx,ny,mxy,nghost);
        hipHostRegister(halo_yz, halo_size, hipHostRegisterDefault);
        hipMemcpyAsync(halo_yz, d_halo_yz, halo_size, hipMemcpyDeviceToHost,strLeftRight);

// unpack on host side

        for (i=0;i<nz-2*nghost;i++)
        {
                for (j=0;j<ny-2*nghost;j++)
                {
                        // inner left plate
                        hipMemcpyAsync(grid+offset,halo_yz+halo_ind,size,hipMemcpyHostToHost,strLeftRight);
                        halo_ind+=nghost;
                        offset+=x_inc;
                        // inner right plate
                        hipMemcpyAsync(grid+offset,halo_yz+halo_ind,size,hipMemcpyHostToHost,strLeftRight);
                        halo_ind+=nghost;
                        offset+=3*nghost;
                }
                offset+=4*mx*nghost;
        }
}
/****************************************************************************************************************/
__global__ void setIxyPlates(float* d_grid, int mx, int mxy, int nz, int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner xy halos

        long start_offset=(mxy+mx+1)*nghost;
        long grid_ind=start_offset + threadIdx.x + threadIdx.y*mx + threadIdx.z*mxy;

        // inner front plate
        d_grid[grid_ind] = (float) (-grid_ind-1);

        // inner back plate
        grid_ind += (nz-nghost)*mxy;
        d_grid[grid_ind] = (float) (-grid_ind-1);
}
/****************************************************************************************************************/
__global__ void setIxzPlates(float* d_grid, int mx, int mxy, int ny, int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner xz halos


        long start_offset=(2*mxy+mx+1)*nghost;
        long grid_ind=start_offset + threadIdx.x + threadIdx.y*mx + threadIdx.z*mxy;

        // inner bottom plate
        d_grid[grid_ind] = (float) (-grid_ind-1);

        // inner top plate
        grid_ind += (ny-nghost)*mx;
        d_grid[grid_ind] = (float) (-grid_ind-1);
}
/****************************************************************************************/
__global__ void setIyzPlates(float* d_grid,int mx,int nx,int mxy,int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner yz halos

        const long start_offset=((mxy+mx)*2+1)*nghost;

        long grid_ind=start_offset + threadIdx.z*mxy + threadIdx.y*mx + threadIdx.x;
        d_grid[grid_ind] = (float)(-grid_ind-1);
        
        grid_ind+=nx-nghost;
        d_grid[grid_ind] = (float)(-grid_ind-1);
}
/****************************************************************************************************************/
__host__ void copyInnerHalos(float* grid, float* d_grid)
{
//  copies all inner halos from device to host

/* for testing: sets elements of inner halo to their negative linear index -1.

        dim3 threadsxy(nx,ny,nghost);
        setIxyPlates<<<1,threadsxy>>>(d_grid, mx, mxy, nz, nghost);

        dim3 threadsxz(nx,nghost,nz-2*nghost);
        setIxzPlates<<<1,threadsxz>>>(d_grid, mx, mxy, ny, nghost);

        dim3 threadsyz(nghost,ny-2*nghost,nz-2*nghost);
        setIyzPlates<<<1,threadsyz>>>(d_grid, mx, nx, mxy, nghost);
*/
        copyIxyPlates(grid, d_grid);
        copyIxzPlates(grid, d_grid);
        copyIyzPlates(grid, d_grid);
        unlockHostMemInner(grid, d_grid);
}
/****************************************************************************************************************/
__host__ void copyAll(float* grid, float* d_grid)
{
// copies the full data cube from host to device.

 	int size=mxy*mz*sizeof(float);
	hipHostRegister(grid,size,hipHostRegisterDefault);
	hipMemcpy(d_grid, grid, size, hipMemcpyHostToDevice);
	hipHostUnregister(grid);
}
/****************************************************************************************************************/
__host__ void copyInnerAll(float* grid, float* d_grid)
{
// copies the full inner data cube from device to host

        size_t px=mx*sizeof(float);
        size_t sx=nx*sizeof(float);
	long offset=mxy*nghost, offset_data;

        hipHostRegister(grid+offset,mxy*nz*sizeof(float),hipHostRegisterDefault);
        offset_data=offset+(mx+1)*nghost;
        for (int nn=0;nn<nz;nn++) {
        	hipMemcpy2DAsync( grid+offset_data, px, d_grid+offset_data, px, sx, ny, hipMemcpyDeviceToHost, strFront);
    		offset_data+=mxy;
	}
        hipStreamSynchronize(strFront);
        hipHostUnregister(grid+offset);
}
/****************************************************************************************************************/
